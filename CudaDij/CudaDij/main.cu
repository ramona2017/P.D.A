#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>


#define nano 1000000L

__global__ void dijkstras(int *a, int *b, int *n)
{
	int i;
	int d[10], p[10], v[10];
	// d stores distnce/cost of each path
	// p stores path taken
	// v stores the nodes already travelled to
	int k, u, s;
	int check = 0;

	// shared memory on cuda device
	__shared__ int temp[20];
	for (i = 0; i < (*n)*(*n); i++)
	{
		temp[i] = a[i];
	}
	check = check + 1;
	__syncthreads();

	// were passing int s -- node from which distances are calculated
	s = threadIdx.x;
	for (i = 0; i<(*n); i++)
	{
		d[i] = temp[s*(*n) + i];
		if (d[i] != 9999)
			p[i] = 1;
		else
			p[i] = 0;
		v[i] = 0;
	}
	p[s] = 0;
	v[s] = 1;
	for (i = 0; i<((*n) - 1); i++)
	{
		// findmin starts here
		int i1, j1, min = 0;
		for (i1 = 0; i1<(*n); i1++)
		{
			if (v[i1] == 0)
			{
				min = i1;
				break;
			}
		}
		for (j1 = min + 1; j1<(*n); j1++)
		{
			if ((v[j1] == 0) && (d[j1]<d[min]))
				min = j1;
		}
		k = min;
		// findmin ends here
		v[k] = 1;
		for (u = 0; u<(*n); u++)
		{
			if ((v[u] == 0) && (temp[k*(*n) + u] != 9999))
			{
				if (d[u]>d[k] + temp[k*(*n) + u])
				{
					d[u] = d[k] + temp[k*(*n) + u];
					p[u] = k;
				}
			}
		}
		//storing output
		int count = 0;
		for (i = (s*(*n)); i< (s + 1) * (*n); i++)
		{
			b[i] = d[count];
			count++;
		}
	}
	*n = check;
}



main()
{
	int *a, *b, *n;
	int *d_a, *d_b, *d_n;
	int i, j, c;
	int check = 0;
	printf("enter the number of vertices.... : ");
	n = (int*)malloc(sizeof(int));
	scanf("%d", n);
	int size = (*n) * (*n) * sizeof(int);

	//allocating device memory
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_n, sizeof(int));

	a = (int*)malloc(size);
	b = (int*)malloc(size);

	
}