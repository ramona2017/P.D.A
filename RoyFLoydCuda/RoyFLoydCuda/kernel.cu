#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#ifndef __HIPCC__ 
#define __HIPCC__
#endif

#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include<time.h>
#include <stdio.h>
#define INF 999
#define n 5

__global__ void floyd(int *a) {
	int k;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	for (k = 0; k < n; k++) {
		__syncthreads();
		if (a[i + k] + a[k + j] < a[i + j]) {
			a[i + j] = a[i + k] + a[k + j];
		}
	}
}

void print(int **a) {
	int i, j;

	for (i = 0; i < n; i++)
		for (j = 0; j < n; j++)
			printf("%d", a[i*n + j]);
}
int main() {
	time_t t;
	int  *d_a;
	int i, j, k;
	int size;
	size = n*n;
	srand((unsigned)time(&t));

	int **h_a = (int**)malloc(n * sizeof(int));

	hipMalloc((void **)&d_a, size);

	for (i = 0; i < n; i++)
		for (j = 0; j < n; j++)
			h_a[i][j] = rand() % 10;

	hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);

	dim3 threadBlock(n, n);


	floyd << <1, threadBlock >> >(d_a);

	hipMemcpy(h_a, d_a, size, hipMemcpyDeviceToHost);

	hipFree(d_a);
	print(h_a);

}
