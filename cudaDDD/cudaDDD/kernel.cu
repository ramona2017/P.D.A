#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#ifndef __HIPCC__ 
#define __HIPCC__
#endif
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include<time.h>
#include <stdio.h>

#define nano 1000000L

__global__ void dijkstras(int *a, int *b, int *n)
{
	int i;
	int d[10], p[10], v[10];// d este costul fiecarei cai ;                     p :Stochează calea luată; v Stochează nodurile deja vizitate
	int k, u, s, check = 0;

	__shared__ int temp[20];
	for (i = 0; i < (*n)*(*n); i++)
	{
		temp[i] = a[i];
	}
	check = check + 1;
	__syncthreads();

	s = threadIdx.x;
	for (i = 0; i<(*n); i++)
	{
		d[i] = temp[s*(*n) + i];
		if (d[i] != 999)
			p[i] = 1;
		else
			p[i] = 0;
		v[i] = 0;
	}
	p[s] = 0;
	v[s] = 1;

	for (i = 0; i<((*n) - 1); i++)
	{

		int i1, j1, min = 0;
		for (i1 = 0; i1<(*n); i1++)
		{
			if (v[i1] == 0)
			{
				min = i1;
				break;
			}
		}
		for (j1 = min + 1; j1<(*n); j1++)
		{
			if ((v[j1] == 0) && (d[j1]<d[min]))
				min = j1;
		}
		k = min;

		v[k] = 1;
		for (u = 0; u<(*n); u++)
		{
			if ((v[u] == 0) && (temp[k*(*n) + u] != 999))
			{
				if (d[u]>d[k] + temp[k*(*n) + u])
				{
					d[u] = d[k] + temp[k*(*n) + u];
					p[u] = k;
				}
			}
		}
		//storing output
		int count = 0;
		for (i = (s*(*n)); i< (s + 1) * (*n); i++)
		{
			b[i] = d[count];
			count++;
		}
	}
	*n = check;
}




void main()
{
	int *a, *b, *n;
	int *d_a, *d_b, *d_n;
	int i, j, c;
	int check = 0;
	printf("Introduceți numărul de noduri.... : ");
	n = (int*)malloc(sizeof(int));
	scanf("%d", n);
	int size = (*n) * (*n) * sizeof(int);

	//allocating device memory
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_n, sizeof(int));

	a = (int*)malloc(size);
	b = (int*)malloc(size);

	check = check + 1;
	for (i = 0; i<(*n); i++)
		for (j = 0; j <= i; j++)
			if (i == j)
				a[(i*(*n) + j)] = 0;
			else
				a[(i*(*n) + j)] = a[(j*(*n) + i)] = 999;

	printf("\nMatricea initiala: \n");
	for (i = 0; i<(*n); i++)
	{
		for (j = 0; j<(*n); j++)
		{
			printf("%d ", a[i*(*n) + j]);
		}
		printf("\n");
	}

	while (1)
	{
		printf("\n Introduceti nodul sursa, nodul destinatie si costul  \n");
		scanf("%d %d %d", &i, &j, &c);
		if (i == -1)
			break;
		a[(i*(*n) + j)] = a[(j*(*n) + i)] = c;
	}

	printf("\n \n");
	for (i = 0; i<(*n); i++)
	{
		for (j = 0; j<(*n); j++)
		{
			printf("%d ", a[i*(*n) + j]);
		}
		printf("\n");
	}

	check = check + 1;
	// copying input matrix to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_n, n, sizeof(int), hipMemcpyHostToDevice);
	check++;

	int N = *n;

	dijkstras <<< 1, N >>>(d_a, d_b, d_n);
	check++;

	// copying result from device to host
	hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost);
	hipMemcpy(n, d_n, sizeof(int), hipMemcpyDeviceToHost);

	check++;
	//  printing result
	printf("Cele mai scurte căi sunt....");
	for (i = 0; i<(N); i++)
	{
		for (j = 0; j<(N); j++)
		{
			if (i != j)
				printf("\n Costul căii de la %d la %d = %d\n", i, j, b[i*(N)+j]);
		}
		printf("\n\n");
	}

	free(a);
	free(b);
	free(n);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_n);
}

